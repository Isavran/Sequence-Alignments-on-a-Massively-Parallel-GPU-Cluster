#include "hip/hip_runtime.h"
/**
 * g_kernel.cu
 *
 *  Created on: Dec 15, 2012
 *      Author: Ibrahim Savran
 *     Referenced ygao's Kernel
 */

#include <stdio.h>
#include "g_interface.h"

#define MAX_LEN 	801
#define MAXL 		800
#define FMAT_LINES 	2

#define INDEX_FM(a,b) 	(((stride+1)*((a)%FMAT_LINES)+(b))*GROUP_SIZE + tid)
#define INDEX_A(a) 		((bid * stride + (stride * groupA * GROUP_SIZE))+(a))
#define INDEX_B(a) 		((a)*GROUP_SIZE+(tid + (stride * groupB * GROUP_SIZE)))

///CONCURRENT_GROUPS
extern "C" __global__

void needlemanWunsch_CU(
		FLOAT_T *adLookup,
		const char* acA,
		const char* acB,
		int gi,					///
		int gj,					///
		int *anLen,
		FLOAT_T *dDistBig,
		int stride,				/// blokdimx=32; y =1 	tready 0
		int threadNum ) {
	int 	tid  = threadIdx.x;
	int 	bid  = blockIdx.x;				/// 
	int 	bgrp = blockIdx.y; 				/// .
	int 	groupA = gi;
	int 	groupB = gj + /*tgrp * threadNum */ + bgrp * threadNum /* *blockDim.y*/;	/// block dim 256 1 1


	int 	nLenA =  anLen[groupA * GROUP_SIZE + bid];
	int 	nLenB =  anLen[groupB * GROUP_SIZE + tid];

	if ( nLenA == 00 ){
		if ( 0 == nLenB )
			dDistBig[ bgrp * GROUP_SIZE * GROUP_SIZE+bid*GROUP_SIZE+tid ] = 0.0;
		else 	dDistBig[ bgrp * GROUP_SIZE * GROUP_SIZE+bid*GROUP_SIZE+tid ] = GAP_PENALTY; ///dDist[bid*GROUP_SIZE+tid] = GAP_PENALTY;
	} else {
		__syncthreads();

#ifdef SHARE_USAGE
		__shared__ int cache_lo[N_BASES*N_BASES];
		__shared__ int cache_hi[N_BASES*N_BASES];
			if (tid < N_BASES*N_BASES ){
				cache_lo[tid] = __double2loint(adLookup[tid]);
				cache_hi[tid] = __double2hiint(adLookup[tid]);
			}
#endif
		FLOAT_T aadFMatrix[MAXL];	
		short 	dDist[MAXL];			
		short j;
			for ( j = 0; j <= nLenB; j++ ){
				aadFMatrix[j] 	= GAP_PENALTY * j; 
				dDist[j]			= 0;
			}

			FLOAT_T laadFM;
			short left;
			for ( short i = 1; i <= nLenA; i++ ){
				laadFM = GAP_PENALTY * i;///0 a yaz	
				char acm = acA[ INDEX_A(i - 1) ];
				char ai = acm;
				left = 0;
				#pragma unroll 1
				for (j = 1; j <= nLenB; j++){
					FLOAT_T myd1, myd2, myd3;
					char bj =	acB[INDEX_B(j - 1)];
///================================================
#ifdef SHARE_USAGE	
					myd1 = aadFMatrix[j-1] + __hiloint2double(cache_hi[ ai*N_BASES + bj],
																cache_lo[ ai*N_BASES + bj]);
#else
					myd1 = aadFMatrix[j-1] + adLookup[ ai*N_BASES + bj];
#endif
					myd3 = aadFMatrix[j] + GAP_PENALTY;	/// up
					myd2 = laadFM + GAP_PENALTY;		/// left
					if( acm == bj ){
						myd2 += (HOMOPOLYMER_PENALTY - GAP_PENALTY);
						myd3 += (HOMOPOLYMER_PENALTY - GAP_PENALTY);
					}

					if (i == nLenA-1 )	/**/	myd2 = laadFM;///aadFMatrix[adr];
					if (j == nLenB-1 )			myd3 = aadFMatrix[j];///
				
					aadFMatrix[j-1] = laadFM;

					acm =  myd1 < myd2? DIAG: LEFT;		///dChoice1 		= fmin(dChoice1, dChoice2);
					if (acm == LEFT) myd1 = myd2;
					acm = myd1 < myd3 ? acm:UP;			///dChoice1 < dChoice2 ? : sAll[0][otid]	= LEFT;
					if (acm ==UP) myd1 = myd3;
					laadFM = myd1;	 

					short now;
					if 	(acm == DIAG) { 
						now = dDist[j-1]+1;
						acm = ai;
						if(ai != bj)		acm = -1;
						///else				acm = -1;
					}
					else if (acm == LEFT)  { acm = bj; now = left+1; 		if (i==nLenA-1)  now--;}
					else /*if (acm == UP)*/{ acm = ai; now = dDist[j]+1; 	if (j==nLenB-1)  now--;} 
					dDist[j-1] 	= left;
					left		= now;

///-------------------------------------------------
			}	///for j
			aadFMatrix[j-1] = laadFM;
			dDist[j-1]=left;
		}	/// for i
	dDistBig[ bgrp * GROUP_SIZE * GROUP_SIZE+bid * GROUP_SIZE+tid ] = laadFM/((FLOAT_T) left);
	}
}///End of the Kernel

